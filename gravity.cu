
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <signal.h>

// There are ways to get this data but I'm too lazy
#define CUDA_CORES 384

//#define N 7 
#define N 1000 

#define ITERATIONS 864000 

// http://www.wolframalpha.com/input/?i=gravitational+constant+in+km%5E3%2F%28Yg+*+s%5E2%29
#define GRAVITATIONAL_CONSTANT 66.7 // km^3 / (Yg * s^2)
//#define GRAVITATIONAL_CONSTANT 240300.0 // km^3 / (Yg * min^2)
#define TIME_STEP 0.1 

#define SAVE_STEP 100

#define SMOOTHING_SQ 4e3

volatile sig_atomic_t kill_flag = 0; // if the program gets killed, flag for the main loop
void set_kill_flag(int sig){ // can be called asynchronously
          kill_flag = 1; // set flag
}

void random_ints(int* a, int num) {
        int i;
        for(i = 0; i < num; ++i) {
                a[i] = rand();
        }
} 

void random_doubles(double* a, int num, double multiplier) {
        int i;
        for(i = 0; i < num; i++) {
                a[i] = (double)rand() / (double)RAND_MAX * multiplier;
        }
}

void random_double4s(double4* a, int num, double m0, double m1, double m2, double m3) {
        int i;
        for(i = 0; i < num; i++) {
                a[i].x = ((double)rand() / (double)RAND_MAX - 0.5) * m0;
                a[i].y = ((double)rand() / (double)RAND_MAX - 0.5) * m1;
                a[i].z = ((double)rand() / (double)RAND_MAX - 0.5) * m2;
                a[i].w = ((double)rand() / (double)RAND_MAX) * m3;
        }
}

void load_initial_data(double4 *in_pos, double4 *in_vel, int num_particles) {
        FILE *ifp;
        char *mode = "r";

        ifp = fopen("input.csv", mode);

        double w, x, y, z, xv, yv, zv;

        if(ifp == NULL) fprintf(stderr, "OH NO! No file!\n");

        for(int i = 0; i < num_particles; i++) {
                fscanf(ifp, "%lf, %lf, %lf, %lf, %lf, %lf, %lf", &w, &x, &y, &z, &xv, &yv, &zv);
                
                in_pos[i].w = w;
                in_pos[i].x = x;
                in_pos[i].y = y;
                in_pos[i].z = z;

                in_vel[i].w = 0.0;
                in_vel[i].x = xv;
                in_vel[i].y = yv;
                in_vel[i].z = zv;

                printf("%g, %g, %g, %g, %g, %g, %g\n", w, x, y,z, xv, yv, zv);
        }
        fclose(ifp);
}

void save_continue_csv(const char *filename, double4 *poss, double4 *vels) {
        FILE *next_input = fopen(filename, "w");

        for(int j = 0; j < N; j++)
        fprintf(next_input, "%g,%g,%g,%g,%g,%g,%g\n", poss[j].w, poss[j].x, poss[j].y, poss[j].z, vels[j].x, vels[j].y, vels[j].z);

        fclose(next_input);
        printf("Saved.");
}


__device__ double3 interaction(double4 body_a, double4 body_b, double3 accel) {
        double3 r;
        r.x = body_b.x - body_a.x;
        r.y = body_b.y - body_a.y;
        r.z = body_b.z - body_a.z;
 
        double dist_sq = r.x * r.x + r.y * r.y + r.z * r.z + SMOOTHING_SQ;
 
        //dist_sq += 4e6; // softening factor
 
        double inv_dist = rsqrt(dist_sq);
        double inv_dist_cube = inv_dist * inv_dist * inv_dist;
 
        double accel_total = GRAVITATIONAL_CONSTANT * body_b.w * inv_dist_cube;
 
        accel.x += r.x * accel_total;
        accel.y += r.y * accel_total;
        accel.z += r.z * accel_total;
        
        return accel;
}

__device__ double3 tile_calculation(double4 body_a, double3 accel) {
        int i;
        extern __shared__ double4 shared_positions[];
        //__shared__ double4 shared_positions[N];
        //double4 *shared_positions = SharedMemory();


#pragma unroll 128
        for(i = 0; i < blockDim.x; i++) {
                accel = interaction(body_a, shared_positions[i], accel);
        }

        return accel;
}

__device__ double4 calculate_accel(double4 *positions, int num_tiles, int num_particles) {
        extern __shared__ double4 shared_positions[];

        double4 cur_body; // current block's body

        int tile;

        double3 accel = {0.0, 0.0, 0.0};

        int gtid = blockIdx.x * blockDim.x + threadIdx.x;


        cur_body = positions[gtid];

        for(tile = 0; tile < num_tiles; tile++) {
                int idx = tile * blockDim.x + threadIdx.x;
                shared_positions[threadIdx.x] = positions[idx];
                __syncthreads();
#pragma unroll 128
                for(int counter = 0; counter < blockDim.x; counter++) {
                        accel = interaction(cur_body, shared_positions[counter], accel);
                }
                __syncthreads();
        }
        

        double4 accel4 = {accel.x, accel.y, accel.z, 0.0};
        return accel4;
}

__global__ void integrate(double4 *positions, double4 *vels, int num_tiles, int num_particles) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

        if(index >= num_particles) {
                return;
        }

        double4 position = positions[index];

        double4 accel = calculate_accel(positions, num_tiles, num_particles);
        
        double4 velocity = vels[index]; 

        velocity.x += accel.x * TIME_STEP;
        velocity.y += accel.y * TIME_STEP;
        velocity.z += accel.z * TIME_STEP;

        position.x += velocity.x * TIME_STEP;
        position.y += velocity.y * TIME_STEP;
        position.z += velocity.z * TIME_STEP;

        __syncthreads();

        positions[index] = position;
        vels[index] = velocity;
}


int main(int argc, char *argv[]) {
        signal(SIGINT, set_kill_flag);
        
        int num_particles = N;
        int block_size = num_particles;

        int num_blocks = (num_particles + block_size-1) / block_size;
        int num_tiles = (num_particles + block_size - 1) / block_size;
        int shared_mem_size = block_size * 4 * sizeof(double); // 4 floats for pos

        double4 *positions, *vels;
        double4 *dev_positions, *dev_vels;

        int size = N * sizeof(double4);
        
        hipMalloc((void**)&dev_positions, size);
        hipMalloc((void**)&dev_vels, size);
         
        positions = (double4*)malloc(size);
        vels = (double4*)malloc(size);


        //int seed = time(NULL);
        //srand(seed);
        //random_double4s(positions, N, 6e8, 6e8, 6e3, 11.6 * 2.0);
        //random_double4s(vels, N, 0.5e2, 0.5e2, 0.1, 0.0);

        load_initial_data(positions, vels, N);

        hipMemcpy(dev_positions, positions, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_vels, vels, size, hipMemcpyHostToDevice);


        FILE *fp = fopen("output.csv", "w");

        for(int i = 0; i < ITERATIONS; i++) {

                integrate<<<num_blocks, block_size, shared_mem_size>>>(dev_positions, dev_vels, num_tiles, num_particles);

                hipMemcpy(positions, dev_positions, size, hipMemcpyDeviceToHost);
                hipMemcpy(vels, dev_vels, size, hipMemcpyDeviceToHost);

                if(i % SAVE_STEP == 0) {
                        printf("%.2f\n", (double)i * 100.0 / (double)ITERATIONS);
                        for(int j = 0; j < N; j++)
                        fprintf(fp, "%g,%g,%g,%g,%g,%g\n", positions[j].x, positions[j].y, positions[j].z, vels[j].x, vels[j].y, vels[j].z);
                }
                if(kill_flag) {
                        break;
                }
        }
        fclose(fp);

        if(kill_flag) {
                save_continue_csv("recovered-input.csv", positions, vels);                    
        } else {
                save_continue_csv("next-input.csv", positions, vels);                    
        }

        hipFree(dev_positions);
        hipFree(dev_vels);
        
        free(positions); free(vels);

        return 0;
}

