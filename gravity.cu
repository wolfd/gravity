#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 512 
#define THREADS_PER_BLOCK 512 

#define ITERATIONS 300

//#define GRAVITATIONAL_CONSTANT 66.7 // km^3 / (Yg * s^2)
#define GRAVITATIONAL_CONSTANT 240300.0 // km^3 / (Yg * min^2)
#define TIME_STEP 60.0 //
// http://www.wolframalpha.com/input/?i=gravitational+constant+in+km%5E3%2F%28Yg+*+s%5E2%29

__constant__ double G;

void random_ints(int* a, int num) {
        int i;
        for(i = 0; i < num; ++i) {
                a[i] = rand();
        //        a[i] = 1;
        }
} 

void random_doubles(double* a, int num, double multiplier) {
        int i;
        for(i = 0; i < num; i++) {
                a[i] = (double)rand() / (double)RAND_MAX * multiplier;
        }
}

__global__ void add(int *a, int *b, int *c) {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        c[index] = a[index] + b[index];
        //*c = *a + *b;
}

__global__ void dot(int *a, int *b, int *c) {
        __shared__ int temp[THREADS_PER_BLOCK];

        int index = threadIdx.x + blockIdx.x * blockDim.x;

        temp[threadIdx.x] = a[index] * b[index];

        __syncthreads();

        if(0 == threadIdx.x) {
                int sum = 0;
                for(int i = 0; i < THREADS_PER_BLOCK; i++) {
                        sum += temp[i];
                }
                atomicAdd(c, sum);
        }
}

__global__ void update_positions(double *x, double *y, double *vx, double *vy) {
        x[blockIdx.x] += vx[blockIdx.x] * TIME_STEP;
        y[blockIdx.x] += vy[blockIdx.x] * TIME_STEP;
        //if(12 == blockIdx.x)
        //printf("%g, %g, %d\n", vx[blockIdx.x] * TIME_STEP, vx[blockIdx.x], blockIdx.x);
        //vx[blockIdx.x] = 0.0;
        //vy[blockIdx.x] = 0.0;
}

__global__ void gravity(double *x, double *y, double *m, double *vx, double *vy) {
        __shared__ double ax[THREADS_PER_BLOCK];
        __shared__ double ay[THREADS_PER_BLOCK];
        // Initialize this thread's values to 0
        ax[threadIdx.x] = 0.0;
        ay[threadIdx.x] = 0.0;

        double d_x, d_y, accel_part, dist_sq;

        if(blockIdx.x != threadIdx.x) {
                d_x = x[blockIdx.x] - x[threadIdx.x];
                d_y = y[blockIdx.x] - y[threadIdx.x];

                dist_sq = d_x * d_x + d_y * d_y; 
                accel_part = GRAVITATIONAL_CONSTANT * m[threadIdx.x] / dist_sq;
                if(isnan(accel_part) || isinf(accel_part)) {
                        ax[threadIdx.x] = 0.0;
                        ay[threadIdx.x] = 0.0;
                } else {
                        double dist = sqrt(dist_sq);
                        //if(accel_part > 1e-17)
                        //printf("%g, %g\n", accel_part * (d_x / dist), dist);
                        if(dist_sq < 1000 || isnan(accel_part) || isnan(dist)) {
                                ax[threadIdx.x] = 0.0;
                                ay[threadIdx.x] = 0.0;
                        } else {
                                ax[threadIdx.x] = - accel_part * (d_x / dist);
                                ay[threadIdx.x] = - accel_part * (d_y / dist);
                        }
                }
        } else {
                ax[threadIdx.x] = 0.0;
                ay[threadIdx.x] = 0.0;
        }

        /*
           G * M * m
             -----
            dist^2
        */

        __syncthreads();

        if(0 == threadIdx.x) {

                for(int i = 0; i < blockDim.x; i++) {
                        if(i != blockIdx.x && !isnan(ax[i]) && !isnan(ay[i])) {
                                vx[blockIdx.x] += ax[i];
                                vy[blockIdx.x] += ay[i];
                        }
                }
        }
}

int main(void) {
        double *x, *y, *m, *vx, *vy;
        double *dev_x, *dev_y, *dev_m, *dev_vx, *dev_vy;
        int size = N * sizeof(double);

        /*hipMemcpyToSymbol(HIP_SYMBOL(G),
                        GRAVITATIONAL_CONSTANT,
                        sizeof(double),
                        0,
                        hipMemcpyHostToDevice);
        */

        hipMalloc((void**)&dev_x, size);
        hipMalloc((void**)&dev_y, size);
        hipMalloc((void**)&dev_m, size);
        hipMalloc((void**)&dev_vx, size);
        hipMalloc((void**)&dev_vy, size);

        x = (double*)malloc(size);
        y = (double*)malloc(size);
        m = (double*)malloc(size);
        vx = (double*)malloc(size);
        vy = (double*)malloc(size);

        //memset(vx, 0, size);
        //memset(vy, 0, size);
        

        int seed = time(NULL);
        srand(seed);

        random_doubles(x, N, 6e5);
        random_doubles(y, N, 6e5);
        random_doubles(m, N, 11.6 * 2.0);
        random_doubles(vx, N, 0.0);
        random_doubles(vy, N, 0.0);

        hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_y, y, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_m, m, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_vx, vx, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_vy, vy, size, hipMemcpyHostToDevice);

        FILE *fp = fopen("locations.csv", "w");


        for(int i = 0; i < ITERATIONS; i++) {
                update_positions<<<N, 1>>>(dev_x, dev_y, dev_vx, dev_vy);
                gravity<<<N, THREADS_PER_BLOCK>>>(dev_x, dev_y, dev_m, dev_vx, dev_vy);

                hipMemcpy(x, dev_x, size, hipMemcpyDeviceToHost);
                hipMemcpy(y, dev_y, size, hipMemcpyDeviceToHost);
                hipMemcpy(vx, dev_vx, size, hipMemcpyDeviceToHost);
                hipMemcpy(vy, dev_vy, size, hipMemcpyDeviceToHost);

                printf("%g\n", (double)i / (double)ITERATIONS);

                for(int j = 0; j < N; j++)
                fprintf(fp, "%d, %d, %g, %g, %g, %g\n", i, j, x[j], y[j], vx[j], vy[j]);
        }

        fclose(fp);

        hipFree(dev_x);
        hipFree(dev_y);
        hipFree(dev_m);
        hipFree(dev_vx);
        hipFree(dev_vy);
        
        free(x); free(y); free(m);

        //printf("Numbers:\n");
        //for(int i = 0; i < N; i++) {
        //        printf("%d: (%g, %g)\n", i, vx[i], vy[i]);
        //}
        
        free(vx); free(vy);

        return 0;
}

