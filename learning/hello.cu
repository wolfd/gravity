
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (2048) 
#define THREADS_PER_BLOCK 512 

void random_ints(int* a, int num) {
        int i;
        for(i = 0; i < num; ++i) {
                a[i] = rand();
        //        a[i] = 1;
        }
} 

__global__ void add(int *a, int *b, int *c) {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        c[index] = a[index] + b[index];
        //*c = *a + *b;
}

__global__ void dot(int *a, int *b, int *c) {
        __shared__ int temp[THREADS_PER_BLOCK];

        int index = threadIdx.x + blockIdx.x * blockDim.x;

        temp[threadIdx.x] = a[index] * b[index];

        __syncthreads();

        if(0 == threadIdx.x) {
                int sum = 0;
                for(int i = 0; i < THREADS_PER_BLOCK; i++) {
                        sum += temp[i];
                }
                atomicAdd(c, sum);
        }
}


int main(void) {
        int *a, *b, *c;
        int *dev_a, *dev_b, *dev_c;
        int size = N * sizeof(int);

        hipMalloc((void**)&dev_a, size);
        hipMalloc((void**)&dev_b, size);
        hipMalloc((void**)&dev_c, sizeof(int));

        a = (int*)malloc(size);
        b = (int*)malloc(size);
        c = (int*)malloc(sizeof(int));

        random_ints(a, N);
        random_ints(b, N);

        hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

        dot<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_c);

        hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
        
        free(a); free(b); 

        printf("Numbers:\n");
        for(int i = 0; i < N; i++) {
                printf("%d\n", c[i]);
        }

        free(c);
        return 0;
}



